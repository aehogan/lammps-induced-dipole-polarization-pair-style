
#include <hip/hip_runtime.h>
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator 

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov 

   See the README file in the top-level LAMMPS directory. 

   ----------------------------------------------------------------------- 

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/ 

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany 

   See the README file in the USER-CUDA directory. 

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#ifdef CUDA_USE_BINNING
#include <stdio.h>
#define MY_PREFIX binning
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "binning_cu.h"
#include "binning_kernel.cu"

void Cuda_PreBinning(cuda_shared_data* sdata)
{
	// initialize only on first call
	short init = 0;
	if(! init)
	{
		init = 1;
		int cuda_dummy_type = sdata->atom.ntypes + 1;
		X_FLOAT outside[3] =
		{
			(sdata->domain.subhi[0] - sdata->domain.sublo[0])/1000.0,
			(sdata->domain.subhi[1] - sdata->domain.sublo[1])/1000.0,
			(sdata->domain.subhi[2] - sdata->domain.sublo[2])/1000.0
		};
		cudaMemcpyToSymbol("binned_size_all"    , & sdata->atom.binned_type.dim[0]  , sizeof(unsigned) );
		cudaMemcpyToSymbol("cuda_dummy_type"    , & cuda_dummy_type                 , sizeof(int)      );
		cudaMemcpyToSymbol("outside"            , & outside                         , sizeof(X_FLOAT)*3);
		cudaMemcpyToSymbol(MY_CONST(binned_type), & sdata->atom.binned_type.dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(binned_x)   , & sdata->atom.binned_x   .dev_data, sizeof(X_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_tag) , & sdata->atom.binned_tag .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(subhi)      ,   sdata->domain.subhi             , sizeof(X_FLOAT)*3);
		// bin_nmax == blockDim.x
		
		// printf("# CUDA: MY_CONST(binned_type) = %s\n", MY_CONST(binned_type));
		// int* p = pre_binning_binned_type; // pre_binning_binned_type is defined here!!
	}
	
	dim3 grid(sdata->domain.bin_dim[0], sdata->domain.bin_dim[1] * sdata->domain.bin_dim[2], 1);
	dim3 threads(sdata->domain.bin_nmax, 1, 1);
	
	MYDBG(printf("# CUDA: Cuda_PreBinning: pre binning grid = (%u, %u, %u)\n", grid.x, grid.y, grid.z);)
	MYDBG(printf("# CUDA: Cuda_PreBinning: pre binning threads = (%u, %u, %u)\n", threads.x, threads.y, threads.z);	)
	PreBinning_Kernel<<<grid, threads>>> ();
	cudaThreadSynchronize();
    MYDBG(printf("ERROR-CUDA pre_binning: %s\n",cudaGetErrorString(cudaGetLastError())));
	CUT_CHECK_ERROR("Cuda_PreBinning: binning Kernel execution failed");
}

void Cuda_Binning(cuda_shared_data* sdata)
{
	MYDBG(	// check assumption in debug mode
		if(sdata->atom.x.dim[1] != 3)
		{
			printf("# CUDA: Cuda_Binning: binning error: atom array dimensions not Nx3\n");
			return;
		}
	)
	
	// initialize only on first call
	short init = 0;
	if(! init)
	{
		init = 0;
		X_FLOAT const_rez_bin_size[3] = 
		{
			(1.0 * sdata->domain.bin_dim[0]-4.0) / (sdata->domain.subhi[0] - sdata->domain.sublo[0]),
			(1.0 * sdata->domain.bin_dim[1]-4.0) / (sdata->domain.subhi[1] - sdata->domain.sublo[1]),
			(1.0 * sdata->domain.bin_dim[2]-4.0) / (sdata->domain.subhi[2] - sdata->domain.sublo[2])
		};
		cudaMemcpyToSymbol("bin_error_count"        , & sdata->atom.bin_error_count.dev_data, sizeof(X_FLOAT)*1);
		cudaMemcpyToSymbol("rez_bin_size"           , & const_rez_bin_size                  , sizeof(X_FLOAT)*3);
		cudaMemcpyToSymbol(MY_CONST(bin_count_all)  , & sdata->atom.bin_count_all  .dev_data, sizeof(unsigned*));
		cudaMemcpyToSymbol(MY_CONST(bin_count_local), & sdata->atom.bin_count_local.dev_data, sizeof(unsigned*));
		cudaMemcpyToSymbol(MY_CONST(bin_dim)        ,   sdata->domain.bin_dim               , sizeof(int3)     );
		cudaMemcpyToSymbol(MY_CONST(bin_nmax)       , & sdata->domain.bin_nmax              , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(binned_f)       , & sdata->atom.binned_f       .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_q)       , & sdata->atom.binned_q       .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_rmass)   , & sdata->atom.binned_rmass   .dev_data, sizeof(V_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_tag)     , & sdata->atom.binned_tag     .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(binned_type)    , & sdata->atom.binned_type    .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(binned_v)       , & sdata->atom.binned_v       .dev_data, sizeof(V_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binpos)         , & sdata->atom.binpos         .dev_data, sizeof(int*));
		cudaMemcpyToSymbol(MY_CONST(f)              , & sdata->atom.f              .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(natoms)         , & sdata->atom.nall                    , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(nghost)         , & sdata->atom.nghost                  , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(nlocal)         , & sdata->atom.nlocal                  , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(nmax)           , & sdata->atom.nmax                    , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(q)              , & sdata->atom.q              .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(rmass)          , & sdata->atom.rmass          .dev_data, sizeof(V_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(sublo)          ,   sdata->domain.sublo                 , sizeof(X_FLOAT)*3);
		cudaMemcpyToSymbol(MY_CONST(tag)            , & sdata->atom.tag            .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(type)           , & sdata->atom.type           .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(v)              , & sdata->atom.v              .dev_data, sizeof(V_FLOAT*) );
	}
	
	dim3 grid((unsigned)(1 + sdata->atom.nlocal/64.0), 1, 1);
	MYDBG( printf("# CUDA: Cuda_Binning: grid dim.x = %u (nlocal: %i)\n", grid.x,sdata->atom.nlocal); )
	dim3 threads(64, 1, 1);
	
	cudaMemset((int*) (sdata->atom.bin_count_all.dev_data),0,sizeof(int)*(sdata->domain.bin_dim[0])*(sdata->domain.bin_dim[1])*(sdata->domain.bin_dim[2]));
	cudaMemset((int*) (sdata->atom.bin_count_local.dev_data),0,sizeof(int)*(sdata->domain.bin_dim[0])*(sdata->domain.bin_dim[1])*(sdata->domain.bin_dim[2]));
	cudaMemset(sdata->atom.bin_error_count.dev_data,0,sizeof(int)*1);
	int binning_error_l[1];
	
	
	Binning_Kernel<<<grid, threads>>> (
		(X_FLOAT*) (sdata->atom.       x.dev_data),
		(X_FLOAT*) (sdata->atom.binned_x.dev_data),
		sdata->atom.q_flag,
		0,
		sdata->atom.rmass_flag
	);
	cudaThreadSynchronize();
	cudaMemcpy((void*) binning_error_l,(void*) sdata->atom.bin_error_count.dev_data,1*sizeof(int),cudaMemcpyDeviceToHost);
	if(binning_error_l[0]!=0) 
	{
		printf("CUDA-ERROR: binning local: could not bin %i atoms\n",binning_error_l[0]);
	}
	CUT_CHECK_ERROR("Cuda_Binning: binning Kernel execution failed");
	
	grid.x=(unsigned)(1 + (sdata->atom.nall-sdata->atom.nlocal)/32.0);
	MYDBG( printf("# CUDA: Cuda_Binning Ghost: grid dim.x = %u\n", grid.x); )
	
	
	Binning_Kernel<<<grid, threads>>> (
		(X_FLOAT*) (sdata->atom.       x.dev_data),
		(X_FLOAT*) (sdata->atom.binned_x.dev_data),
		sdata->atom.q_flag,
		sdata->atom.nlocal,
		sdata->atom.rmass_flag
	);
	cudaThreadSynchronize();
	cudaMemcpy((void*) binning_error_l,(void*) sdata->atom.bin_error_count.dev_data,1*sizeof(int),cudaMemcpyDeviceToHost);
	if(binning_error_l[0]!=0) printf("CUDA-ERROR: binning ghost: could not bin %i atoms\n",binning_error_l[0]);
}

void Cuda_ReverseBinning(cuda_shared_data* sdata)
{
	// initialize only on first call
	short init = 0;
	if(! init)
	{
		init = 0;
		cudaMemcpyToSymbol(MY_CONST(bin_count_all)  , & sdata->atom.bin_count_all  .dev_data, sizeof(unsigned*));
		cudaMemcpyToSymbol(MY_CONST(bin_count_local), & sdata->atom.bin_count_local.dev_data, sizeof(unsigned*));
		cudaMemcpyToSymbol(MY_CONST(bin_dim)        ,   sdata->domain.bin_dim               , sizeof(int3)     );
		cudaMemcpyToSymbol(MY_CONST(binned_f)       , & sdata->atom.binned_f       .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_q)       , & sdata->atom.binned_q       .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(binned_tag)     , & sdata->atom.binned_tag     .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(binned_type)    , & sdata->atom.binned_type    .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(binned_v)       , & sdata->atom.binned_v       .dev_data, sizeof(V_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(f)              , & sdata->atom.f              .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(natoms)         , & sdata->atom.nall                    , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(nmax)           , & sdata->atom.nmax                    , sizeof(unsigned) );
		cudaMemcpyToSymbol(MY_CONST(q)              , & sdata->atom.q              .dev_data, sizeof(F_FLOAT*) );
		cudaMemcpyToSymbol(MY_CONST(tag)            , & sdata->atom.tag            .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(type)           , & sdata->atom.type           .dev_data, sizeof(int*)     );
		cudaMemcpyToSymbol(MY_CONST(v)              , & sdata->atom.v              .dev_data, sizeof(V_FLOAT*) );
	}
	
	dim3 grid((unsigned)(1 + sdata->atom.nlocal/32.0), 1, 1);
	MYDBG( printf("# CUDA: Cuda_ReverseBinning: grid dim.x = %u (nlocal: %i)\n", grid.x,sdata->atom.nlocal); )
	dim3 threads(32, 1, 1);

	ReverseBinning_Kernel<<<grid, threads>>> (
		(X_FLOAT*) (sdata->atom.       x.dev_data),
		(X_FLOAT*) (sdata->atom.binned_x.dev_data),
		sdata->atom.q_flag
	);
	cudaThreadSynchronize();
	CUT_CHECK_ERROR("Cuda_Binning: reverse binning Kernel execution failed");
}

#endif
