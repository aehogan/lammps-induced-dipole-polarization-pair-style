#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator 

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov 

   See the README file in the top-level LAMMPS directory. 

   ----------------------------------------------------------------------- 

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/ 

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany 

   See the README file in the USER-CUDA directory. 

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

// load some variables from shared cuda data into device's constant memory:
__device__ __constant__ X_FLOAT rez_bin_size[3];
__device__ __constant__ unsigned* bin_error_count;

__device__ __constant__ int cuda_dummy_type;
__device__ __constant__ unsigned binned_size_all;
__device__ __constant__ X_FLOAT outside[3];

__global__ void PreBinning_Kernel()
{
	const unsigned bin = gridDim.y * blockIdx.x + blockIdx.y;
	
	if(bin < gridDim.x * gridDim.y) // TODO: suspected always to be true
	{
		_binned_type[blockDim.x * bin + threadIdx.x] = cuda_dummy_type;
		
		const int i = 3*blockDim.x * bin + threadIdx.x;
		X_FLOAT* binned_x = _binned_x + i; *binned_x = _subhi[0] + outside[0] * (1+i);
		binned_x += blockDim.x;            *binned_x = _subhi[1] + outside[1] * (1+i);
		binned_x += blockDim.x;            *binned_x = _subhi[2] + outside[2] * (1+i);
		_binned_tag[i]=-1;
	}
}

__global__ void Binning_Kernel(X_FLOAT* x, X_FLOAT* binned_x, int q_flag, int offset, int rmass_flag)
{
	const unsigned i = blockDim.x * blockIdx.x + threadIdx.x+offset;
	
	int binatoms=_natoms;
	if(offset==0) binatoms=_nlocal ;

	if(i < binatoms)
	{
		// copy atom position from global device memory to local register
		// in this 3 steps to get as much coalesced access as possible
		X_FLOAT my_xX, my_xY, my_xZ;
		x += i;        my_xX = *x;
		x += _nmax;  my_xY = *x;
		x += _nmax;  my_xZ = *x;
		//my_xX=x[i];
		//my_xY=x[i+_nmax];
		//my_xZ=x[i+2*_nmax];
		
		
		// calculate flat bin index
		int bx=__float2int_rd(rez_bin_size[0] * (my_xX - _sublo[0]))+2;
		int by=__float2int_rd(rez_bin_size[1] * (my_xY - _sublo[1]))+2;
		int bz=__float2int_rd(rez_bin_size[2] * (my_xZ - _sublo[2]))+2;

		bx-=bx*negativCUDA(1.0f*bx);
		bx-=(bx-_bin_dim.x+1)*negativCUDA(1.0f*_bin_dim.x-1.0f-1.0f*bx);
		by-=by*negativCUDA(1.0f*by);
		by-=(by-_bin_dim.y+1)*negativCUDA(1.0f*_bin_dim.y-1.0f-1.0f*by);
		bz-=bz*negativCUDA(1.0f*bz);
		bz-=(bz-_bin_dim.z+1)*negativCUDA(1.0f*_bin_dim.z-1.0f-1.0f*bz);
		

		const unsigned j = _bin_dim.z * ( _bin_dim.y *bx+by)+bz;
		
		// add new atom to bin, get bin-array position
		const unsigned k = atomicAdd(& _bin_count_all[j], 1);
		if(offset==0) atomicAdd(& _bin_count_local[j], 1);
		if(k < _bin_nmax)
		{
			// copy register values back to global device memory
			unsigned pos = 3*_bin_nmax * j + k;
			_binpos[i]=pos;
			binned_x += pos;       *binned_x = my_xX;
			binned_x += _bin_nmax; *binned_x = my_xY;
			binned_x += _bin_nmax; *binned_x = my_xZ;
			
			// also copy velocity and force accordingly
		
			binned_x  = _binned_v + pos; x  = _v + i;  *binned_x = *x;
			binned_x += _bin_nmax;       x += _nmax;   *binned_x = *x;
			binned_x += _bin_nmax;       x += _nmax;   *binned_x = *x;
			
			binned_x  = _binned_f + pos; x  = _f + i;  *binned_x = *x;
			binned_x += _bin_nmax;       x += _nmax;   *binned_x = *x;
			binned_x += _bin_nmax;       x += _nmax;   *binned_x = *x;
			
			pos = _bin_nmax * j + k;
				_binned_type [pos] = _type[i];
				_binned_tag  [pos] = _tag[i];
			if(rmass_flag)
				_binned_rmass[pos] = _rmass[i];
			if(q_flag)
				_binned_q    [pos] = _q[i];
		}
		else
		{	// normally, this should not happen:
			int errorn=atomicAdd(bin_error_count, 1);
			MYEMUDBG( printf("# CUDA: Binning_Kernel: WARNING: atom %i ignored, no place left in bin %u\n", i, j); )
		}
	}
}

__global__ void ReverseBinning_Kernel(X_FLOAT* x, X_FLOAT* binned_x,int q_flag)
{
	const unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i < _nlocal)
	{
	unsigned bin_pos3 = _binpos[i];
	unsigned bin_pos=bin_pos3/(3*_bin_nmax);
	bin_pos*=_bin_nmax;
	bin_pos+=bin_pos3-bin_pos*3;

		binned_x  = _binned_x + bin_pos3; x  = x + i; *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		
		binned_x  = _binned_v + bin_pos3; x  = _v + i; *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		
		binned_x  = _binned_f + bin_pos3; x  = _f + i; *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		binned_x += _bin_nmax;           x += _nmax;  *x = *binned_x;
		
		
		_type[i] = _binned_type[bin_pos];
		_tag[i] = _binned_tag[bin_pos];
		if(q_flag) _q[i] = _binned_q[bin_pos];
	}
}
