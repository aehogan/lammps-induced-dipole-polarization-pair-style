#include "hip/hip_runtime.h"
// **************************************************************************
//                                coul_dsf.cu
//                             -------------------
//                           Trung Dac Nguyen (ORNL)
//
//  Device code for acceleration of the coul/dsf pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : 8/15/2012
//    email                : nguyentd@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL

#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float> q_tex;
#else
texture<int4,1> pos_tex;
texture<int2> q_tex;
#endif

#else
#define pos_tex x_
#define q_tex q_
#endif

#define MY_PIS (acctyp)1.77245385090551602729

__kernel void k_coul_dsf(__global numtyp4 *x_, const int lj_types, 
                         __global numtyp *sp_lj_in, __global int *dev_nbor, 
                         __global int *dev_packed, __global acctyp4 *ans,
                         __global acctyp *engv, const int eflag,
                         const int vflag, const int inum,
                         const int nbor_pitch, __global numtyp *q_ ,
                         const numtyp cut_coulsq, const numtyp qqrd2e,
                         const numtyp e_shift, const numtyp f_shift, 
                         const numtyp alpha, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);

    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_coul, r, prefactor, erfcc;
      factor_coul = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force;

        r = ucl_sqrt(rsq);
        fetch(prefactor,j,q_tex);
        prefactor *= factor_coul * qqrd2e*qtmp/r;
        numtyp erfcd = ucl_exp(-alpha*alpha*rsq);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*alpha*r);
        erfcc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * erfcd;
        forcecoul = prefactor * (erfcc + 2.0*alpha/MY_PIS*r*erfcd + 
                                 rsq*f_shift);
        
        force = forcecoul * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq) {
            numtyp e=prefactor*(erfcc-r*e_shift-rsq*f_shift);
            e_coul += e;
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

__kernel void k_coul_dsf_fast(__global numtyp4 *x_, __global numtyp* sp_lj_in,
                              __global int *dev_nbor, __global int *dev_packed,
                              __global acctyp4 *ans, __global acctyp *engv, 
                              const int eflag, const int vflag, const int inum, 
                              const int nbor_pitch, __global numtyp *q_,
                              const numtyp cut_coulsq, const numtyp qqrd2e,
                              const numtyp e_shift, const numtyp f_shift, 
                              const numtyp alpha, const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  
  acctyp energy=(acctyp)0;
  acctyp e_coul=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;
  
  __syncthreads();
  
  if (ii<inum) {
    __global int *nbor, *list_end;
    int i, numj, n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);
  
    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp qtmp; fetch(qtmp,i,q_tex);
 
    for ( ; nbor<list_end; nbor+=n_stride) {
      int j=*nbor;

      numtyp factor_coul, r, prefactor, erfcc;
      factor_coul = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq < cut_coulsq) {
        numtyp r2inv=ucl_recip(rsq);
        numtyp forcecoul, force;

        r = ucl_sqrt(rsq);
        fetch(prefactor,j,q_tex);
        prefactor *= factor_coul * qqrd2e*qtmp/r;
        numtyp erfcd = ucl_exp(-alpha*alpha*rsq);
        numtyp t = ucl_recip((numtyp)1.0 + EWALD_P*alpha*r);
        erfcc = t * (A1+t*(A2+t*(A3+t*(A4+t*A5)))) * erfcd;
        forcecoul = prefactor * (erfcc + 2.0*alpha/MY_PIS*r*erfcd + 
                                   rsq*f_shift);
        
        force = forcecoul * r2inv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          if (rsq < cut_coulsq) {
            numtyp e=prefactor*(erfcc-r*e_shift-rsq*f_shift);
            e_coul += e;
          }
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,offset,eflag,
                    vflag,ans,engv);
  } // if ii
}

